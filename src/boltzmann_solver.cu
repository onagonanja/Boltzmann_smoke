#include "hip/hip_runtime.h"
﻿#ifdef _MSC_VER
extern "C" {
    __declspec(dllexport) void __cudaRegisterLinkedBinary_e68baee6_19_boltzmann_solver_cu_c_dx(void) {}
}
#endif

#include "boltzmann_solver.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>
#include <cmath>
#include <hiprand/hiprand_kernel.h>

// D3Q19 model velocity vectors
__constant__ float c_dx[19] = {0, 1,-1, 0, 0, 0, 0, 1,-1, 1,-1, 1,-1, 1,-1, 0, 0, 0, 0};
__constant__ float c_dy[19] = {0, 0, 0, 1,-1, 0, 0, 1, 1,-1,-1, 0, 0, 0, 0, 1,-1, 1,-1};
__constant__ float c_dz[19] = {0, 0, 0, 0, 0, 1,-1, 0, 0, 0, 0, 1, 1,-1,-1, 1, 1,-1,-1};

// Weight coefficients
__constant__ float w[19] = {
    1.0f/3.0f, 1.0f/18.0f, 1.0f/18.0f, 1.0f/18.0f, 1.0f/18.0f, 1.0f/18.0f, 1.0f/18.0f,
    1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f,
    1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f
};

// D3Q7 model constants (for temperature field)
__device__ __constant__ float c_t_dx[7] = {0.0f,  1.0f, -1.0f,  0.0f,  0.0f,  0.0f,  0.0f};
__device__ __constant__ float c_t_dy[7] = {0.0f,  0.0f,  0.0f,  1.0f, -1.0f,  0.0f,  0.0f};
__device__ __constant__ float c_t_dz[7] = {0.0f,  0.0f,  0.0f,  0.0f,  0.0f,  1.0f, -1.0f};
__device__ __constant__ float w_t[7] = {1.0f/4.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f};

// D3Q7 model constants (for vorticity field)
__device__ __constant__ float c_h_dx[7] = {0.0f,  1.0f, -1.0f,  0.0f,  0.0f,  0.0f,  0.0f};
__device__ __constant__ float c_h_dy[7] = {0.0f,  0.0f,  0.0f,  1.0f, -1.0f,  0.0f,  0.0f};
__device__ __constant__ float c_h_dz[7] = {0.0f,  0.0f,  0.0f,  0.0f,  0.0f,  1.0f, -1.0f};
__device__ __constant__ float w_h[7] = {1.0f/4.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f, 1.0f/8.0f};

__device__ int found_negative_f = 0;
__device__ int found_negative_g = 0;

__constant__ int focused_point_x = 0;
__constant__ int focused_point_y = 127;
__constant__ int focused_point_z = 0;


// External force update kernel
__global__ void updateExternalForcesKernel(float* force_x, float* force_y, float* force_z, 
                                          float* temperature, float* vorticity_x, float* vorticity_y, float* vorticity_z,
                                          int nx, int ny, int nz, float beta, float lambda, int current_step) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nx * ny * nz) return;

    // Initialize random number generator
    hiprandState state;
    hiprand_init(clock64(), idx + current_step, 0, &state);

    float T_local = temperature[idx];
    float3 vort_local = make_float3(vorticity_x[idx], vorticity_y[idx], vorticity_z[idx]);

    // Calculate buoyancy force
    const float g = 9.81f;
    const float T_ref = 300.0f;
    float rand_factor = 0.5f + 0.5f * hiprand_uniform(&state);
    float3 F_buoy = make_float3(0.0f, beta * g * (T_local - T_ref) * rand_factor, 0.0f);

    // Calculate vorticity constraint force F_conf = λ|ω - ∇×v|(N × ω)
    float vort_mag = sqrtf(vort_local.x * vort_local.x + vort_local.y * vort_local.y + vort_local.z * vort_local.z);
    
    float3 F_conf = make_float3(0.0f, 0.0f, 0.0f);
    if (vort_mag > 1e-6f) {
        float3 N = make_float3(vort_local.x / vort_mag, vort_local.y / vort_mag, vort_local.z / vort_mag);
        // Simplified constraint force calculation
        float constraint_strength = lambda * vort_mag;
        F_conf.x = constraint_strength * N.x;
        F_conf.y = constraint_strength * N.y;
        F_conf.z = constraint_strength * N.z;
    }

    // Total force
    force_x[idx] = F_buoy.x + F_conf.x;
    force_y[idx] = F_buoy.y + F_conf.y;
    force_z[idx] = F_buoy.z + F_conf.z;
    if (F_conf.y > 0 || F_conf.z > 0 || F_conf.x > 0) {
        // printf("F_conf: %f, %f, %f\n", F_conf.x, F_conf.y, F_conf.z);
    }
}

// Modified fluid collision step without external force calculation
__global__ void fluidCollisionKernel(float* f, float* rho, float* vel_x, float* vel_y, float* vel_z, 
                                    float* tau_f, float* force_x, float* force_y, float* force_z,
                                    int nx, int ny, int nz, int current_step, float velocity_limit, 
                                    float tau_rand_factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nx * ny * nz) return;

    // Initialize random number generator
    hiprandState state;
    hiprand_init(clock64(), idx + current_step, 0, &state);

    float f_eq[19];
    float rho_local = rho[idx];
    float3 vel_local = make_float3(vel_x[idx], vel_y[idx], vel_z[idx]);
    float3 force_local = make_float3(force_x[idx], force_y[idx], force_z[idx]);

    vel_local.y += force_local.y;

    // Limit velocity
    float u_sq = vel_local.x * vel_local.x + vel_local.y * vel_local.y + vel_local.z * vel_local.z;
    float u_mag = sqrtf(u_sq);
    if (u_mag > velocity_limit) {
        float scale = velocity_limit / u_mag;
        vel_local.x *= scale;
        vel_local.y *= scale;
        vel_local.z *= scale;
        u_sq = velocity_limit * velocity_limit;
    }

    // Calculate equilibrium distribution
    for (int i = 0; i < 19; i++) {
        float ci_dot_u = c_dx[i] * vel_local.x + c_dy[i] * vel_local.y + c_dz[i] * vel_local.z;
        f_eq[i] = w[i] * rho_local * (1.0f + 3.0f * ci_dot_u + 4.5f * ci_dot_u * ci_dot_u - 1.5f * u_sq);
    }

    float tau = tau_f[idx] + tau_rand_factor * hiprand_uniform(&state);
    float c_s2 = 1.0f / 3.0f;

    // Collision step with external force
    for (int i = 0; i < 19; i++) {
        float ci_dot_u = c_dx[i] * vel_local.x + c_dy[i] * vel_local.y + c_dz[i] * vel_local.z;
        float F_x = w[i] * ( (c_dx[i] - vel_local.x) / c_s2 + ci_dot_u * c_dx[i] / (c_s2 * c_s2)) * force_local.x;
        float F_y = w[i] * ( (c_dy[i] - vel_local.y) / c_s2 + ci_dot_u * c_dy[i] / (c_s2 * c_s2)) * force_local.y;
        float F_z = w[i] * ( (c_dz[i] - vel_local.z) / c_s2 + ci_dot_u * c_dz[i] / (c_s2 * c_s2)) * force_local.z;
        float Fi = F_x + F_y + F_z;
        f[19*idx + i] = f[19*idx + i] - (1.0f/tau) * (f[19*idx + i] - f_eq[i]) + 0.001f *(1.0f - 0.5f / tau) * Fi;
    }
}

// Temperature field collision step
__global__ void temperatureCollisionKernel(float* g, float* temperature, float* vel_x, float* vel_y, float* vel_z, float* tau_t, int nx, int ny, int nz, float velocity_limit) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nx * ny * nz) return;

    float g_eq[7];
    float T_local = temperature[idx];
    float3 vel_local = make_float3(vel_x[idx], vel_y[idx], vel_z[idx]);

    float u_sq = vel_local.x * vel_local.x + vel_local.y * vel_local.y + vel_local.z * vel_local.z;
    float u_mag = sqrtf(u_sq);
    velocity_limit = velocity_limit * 1.0f;

    // Calculate equilibrium distribution for temperature field
    for (int i = 0; i < 7; i++) {
        float ci_dot_u = c_t_dx[i] * vel_local.x + c_t_dy[i] * vel_local.y + c_t_dz[i] * vel_local.z;
        g_eq[i] = w_t[i] * T_local * (1.0f + 3.0f * ci_dot_u);
    }

    float tau = tau_t[idx];
    for (int i = 0; i < 7; i++) {
        g[7*idx + i] = g[7*idx + i] - (1.0f/tau) * (g[7*idx + i] - g_eq[i]);
    }
}

// Temperature field streaming step with selectable boundary conditions
__global__ void temperatureStreamingKernel(float* g, float* g_new, int nx, int ny, int nz, int bc_type, float dirichlet_temperature) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x >= nx || y >= ny || z >= nz) return;

    int idx = z * nx * ny + y * nx + x;

    for (int i = 0; i < 7; i++) {
        int x_next = x + c_t_dx[i];
        int y_next = y + c_t_dy[i];
        int z_next = z + c_t_dz[i];

        bool is_boundary = (x_next < 0 || x_next >= nx || y_next < 0 || y_next >= ny || z_next < 0 || z_next >= nz);
        if (is_boundary) {
            if (bc_type == 0) { // Adiabatic
                int reflected_i = -1;
                for (int j = 0; j < 7; j++) {
                    if (c_t_dx[j] == -c_t_dx[i] && c_t_dy[j] == -c_t_dy[i] && c_t_dz[j] == -c_t_dz[i]) {
                        reflected_i = j;
                        break;
                    }
                }
                if (reflected_i >= 0) {
                    g_new[7*idx + i] = g[7*idx + reflected_i];
                } else {
                    g_new[7*idx + i] = g[7*idx + i];
                }
            } else if (bc_type == 1) { // Dirichlet
                g_new[7*idx + i] = dirichlet_temperature * w_t[i];
            } else if (bc_type == 2) { // Periodic
                int x_p = (x_next + nx) % nx;
                int y_p = (y_next + ny) % ny;
                int z_p = (z_next + nz) % nz;
                int idx_p = z_p * nx * ny + y_p * nx + x_p;
                g_new[7*idx + i] = g[7*idx_p + i];
            }
        } else {
            int idx_next = z_next * nx * ny + y_next * nx + x_next;
            g_new[7*idx + i] = g[7*idx_next + i];
        }
    }
}

// Temperature field update kernel
__global__ void updateTemperatureKernel(float* g, float* temperature, int nx, int ny, int nz) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nx * ny * nz) return;

    float T = 0.0f;
    for (int i = 0; i < 7; i++) {
        T += g[7*idx + i];
    }
    temperature[idx] = T;
}

// Scalar vorticity update kernel
__global__ void updateScalarVorticityKernel(float* h, float* scalar_vorticity, int nx, int ny, int nz) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nx * ny * nz) return;

    float omega = 0.0f;
    for (int i = 0; i < 7; i++) {
        omega += h[7*idx + i];
    }
    scalar_vorticity[idx] = omega;
}

// CUDA kernel for fluid streaming step
__global__ void streamingKernel(float* f, float* f_new, int nx, int ny, int nz) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= nx || y >= ny || z >= nz) return;
    
    int idx = z * nx * ny + y * nx + x;
    
    // Streaming in each direction
    for (int i = 0; i < 19; i++) {
        int x_new = x + (int)c_dx[i];
        int y_new = y + (int)c_dy[i];
        int z_new = z + (int)c_dz[i];
        
        // Apply open boundary conditions
        if (x_new < 0) x_new = 0;
        if (x_new >= nx) x_new = nx - 1;
        if (y_new < 0) y_new = 0;
        if (y_new >= ny) y_new = ny - 1;
        if (z_new < 0) z_new = 0;
        if (z_new >= nz) z_new = nz - 1;
        
        int idx_new = z_new * nx * ny + y_new * nx + x_new;
        f_new[19 * idx_new + i] = f[19 * idx + i];
    }
}

// CUDA kernel for macroscopic quantity calculation
__global__ void calculateMacroKernel(float* f, float* rho, float* vel_x, float* vel_y, float* vel_z, int nx, int ny, int nz) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= nx || y >= ny || z >= nz) return;
    
    int idx = z * nx * ny + y * nx + x;
    
    float rho_local = 0.0f;
    float3 vel_local = make_float3(0.0f, 0.0f, 0.0f);
    
    for (int i = 0; i < 19; i++) {
        float fi = f[19 * idx + i];
        rho_local += fi;
        vel_local.x += c_dx[i] * fi;
        vel_local.y += c_dy[i] * fi;
        vel_local.z += c_dz[i] * fi;
    }
    
    // Normalize velocity
    if (rho_local > 1e-6f) {
        vel_local.x /= rho_local;
        vel_local.y /= rho_local;
        vel_local.z /= rho_local;
    }

    float u_sq = vel_local.x * vel_local.x + vel_local.y * vel_local.y + vel_local.z * vel_local.z;
    float u_mag = sqrtf(u_sq);

    if (u_mag > 0.0577f) {
        float scale = 0.0577f / u_mag;
        vel_local.x *= scale;
        vel_local.y *= scale;
        vel_local.z *= scale;
    }
    
    // Save results
    rho[idx] = rho_local;
    vel_x[idx] = vel_local.x;
    vel_y[idx] = vel_local.y;
    vel_z[idx] = vel_local.z;
}

// CUDA kernel for vorticity calculation
__global__ void calculateVorticityKernel(float* vel_x, float* vel_y, float* vel_z, 
                                        float* vorticity_x, float* vorticity_y, float* vorticity_z, 
                                        int nx, int ny, int nz) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= nx || y >= ny || z >= nz) return;
    
    int idx = z * nx * ny + y * nx + x;
    
    // Calculate velocity gradients using central differences
    float du_dy = 0.0f, du_dz = 0.0f;
    float dv_dx = 0.0f, dv_dz = 0.0f;
    float dw_dx = 0.0f, dw_dy = 0.0f;
    
    // du/dy and du/dz
    if (y > 0 && y < ny - 1) {
        du_dy = (vel_x[idx + nx] - vel_x[idx - nx]) * 0.5f;
    }
    if (z > 0 && z < nz - 1) {
        du_dz = (vel_x[idx + nx * ny] - vel_x[idx - nx * ny]) * 0.5f;
    }
    
    // dv/dx and dv/dz
    if (x > 0 && x < nx - 1) {
        dv_dx = (vel_y[idx + 1] - vel_y[idx - 1]) * 0.5f;
    }
    if (z > 0 && z < nz - 1) {
        dv_dz = (vel_y[idx + nx * ny] - vel_y[idx - nx * ny]) * 0.5f;
    }
    
    // dw/dx and dw/dy
    if (x > 0 && x < nx - 1) {
        dw_dx = (vel_z[idx + 1] - vel_z[idx - 1]) * 0.5f;
    }
    if (y > 0 && y < ny - 1) {
        dw_dy = (vel_z[idx + nx] - vel_z[idx - nx]) * 0.5f;
    }
    
    // Calculate vorticity components: ω = ∇ × v
    vorticity_x[idx] = dw_dy - dv_dz;  // ωx = ∂w/∂y - ∂v/∂z
    vorticity_y[idx] = du_dz - dw_dx;  // ωy = ∂u/∂z - ∂w/∂x
    vorticity_z[idx] = dv_dx - du_dy;  // ωz = ∂v/∂x - ∂u/∂y
}

// Continuous smoke source injection kernel
__global__ void injectSmokeSourceKernel(float* f, float* g, float* rho, float* temperature, int nx, int ny, int nz, 
                                       float source_radius, float source_density, float source_temperature, 
                                       float injection_rate, int current_step) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= nx || y >= ny || z >= nz) return;
    
    int idx = z * nx * ny + y * nx + x;
    
    // Calculate distance from source center
    int center_x = nx / 2;
    int center_y = ny / 8;
    int center_z = nz / 2;
    
    float dx = x - center_x;
    float dy = y - center_y;
    float dz = z - center_z;
    float dist = sqrtf(dx*dx + dy*dy + dz*dz);
    
    // Check if point is within source radius
    if (dist < source_radius) {
        // Initialize random number generator
        hiprandState state;
        hiprand_init(clock64(), idx + current_step, 0, &state);
        
        // Add smoke density with injection rate
        rho[idx] = injection_rate * (0.8f + 0.2f * hiprand_uniform(&state));

        // Update fluid distribution function
        for (int i = 0; i < 19; i++) {
            f[19*idx + i] = w[i] * rho[idx];
        }
        
        // Add temperature with injection rate
        temperature[idx] = 300 + (source_temperature) * (0.8f + 0.2f * hiprand_uniform(&state));
        
        // Update temperature distribution function
        for (int i = 0; i < 7; i++) {
            g[7*idx + i] = w_t[i] * temperature[idx];
        }
    }
}

// Vorticity distribution function collision step
__global__ void vorticityCollisionKernel(float* h, float* scalar_vorticity, float* vel_x, float* vel_y, float* vel_z, 
                                        float* temperature, float* vorticity_x, float* vorticity_y, float* vorticity_z,
                                        int nx, int ny, int nz, float beta, float lambda) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nx * ny * nz) return;

    float h_eq[7];
    float omega_local = scalar_vorticity[idx];
    float3 vel_local = make_float3(vel_x[idx], vel_y[idx], vel_z[idx]);
    float3 vort_local = make_float3(vorticity_x[idx], vorticity_y[idx], vorticity_z[idx]);
    float T_local = temperature[idx];

    // Calculate equilibrium distribution for vorticity field
    float c_s2 = 1.0f / 3.0f;
    for (int i = 0; i < 7; i++) {
        float ci_dot_u = c_h_dx[i] * vel_local.x + c_h_dy[i] * vel_local.y + c_h_dz[i] * vel_local.z;
        h_eq[i] = w_h[i] * omega_local * (1.0f + ci_dot_u / c_s2);
    }

    // Calculate buoyancy force
    const float g = 9.81f;
    const float T_ref = 300.0f;
    float3 F_buoy = make_float3(0.0f, beta * g * (T_local - T_ref), 0.0f);

    // Calculate vorticity constraint force F_conf = λ|ω - ∇×v|(N × ω)
    float3 curl_v = vort_local; // ∇×v (already calculated)
    float omega_mag = sqrtf(omega_local * omega_local);
    float vort_mag = sqrtf(vort_local.x * vort_local.x + vort_local.y * vort_local.y + vort_local.z * vort_local.z);
    
    float3 F_conf = make_float3(0.0f, 0.0f, 0.0f);
    if (omega_mag > 1e-6f && vort_mag > 1e-6f) {
        float diff = fabsf(omega_mag - vort_mag);
        float3 N = make_float3(vort_local.x / vort_mag, vort_local.y / vort_mag, vort_local.z / vort_mag);
        // N × ω (cross product)
        F_conf.x = N.y * omega_local - N.z * 0.0f;
        F_conf.y = N.z * 0.0f - N.x * omega_local;
        F_conf.z = N.x * 0.0f - N.y * 0.0f;
        
        F_conf.x *= lambda * diff;
        F_conf.y *= lambda * diff;
        F_conf.z *= lambda * diff;
    }

    // Total force
    float3 F_total;
    F_total.x = F_buoy.x + F_conf.x;
    F_total.y = F_buoy.y + F_conf.y;
    F_total.z = F_buoy.z + F_conf.z;

    // Collision step with force term
    float tau = 1.0f; // Relaxation time for vorticity
    for (int i = 0; i < 7; i++) {
        float ci_dot_curl_F = c_h_dx[i] * F_total.x + c_h_dy[i] * F_total.y + c_h_dz[i] * F_total.z;
        float force_term = w_h[i] * ci_dot_curl_F / c_s2;
        h[7*idx + i] = h[7*idx + i] - (1.0f/tau) * (h[7*idx + i] - h_eq[i]) + force_term;
    }
}

// Vorticity distribution function streaming step
__global__ void vorticityStreamingKernel(float* h, float* h_new, int nx, int ny, int nz) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x >= nx || y >= ny || z >= nz) return;

    int idx = z * nx * ny + y * nx + x;

    for (int i = 0; i < 7; i++) {
        int x_next = x + c_h_dx[i];
        int y_next = y + c_h_dy[i];
        int z_next = z + c_h_dz[i];

        // Periodic boundary conditions for vorticity
        int x_p = (x_next + nx) % nx;
        int y_p = (y_next + ny) % ny;
        int z_p = (z_next + nz) % nz;
        int idx_next = z_p * nx * ny + y_p * nx + x_p;
        h_new[7*idx + i] = h[7*idx_next + i];
    }
}

BoltzmannSolver::BoltzmannSolver(int nx, int ny, int nz, const InitParams& params)
    : nx_(nx), ny_(ny), nz_(nz), d_f_distribution(nullptr), d_g_distribution(nullptr), d_density(nullptr), d_velocity_x(nullptr), d_velocity_y(nullptr), d_velocity_z(nullptr), d_temperature(nullptr), d_tau_f(nullptr), d_tau_t(nullptr), h_density(nullptr), h_temperature(nullptr), init_params_(params) {
    allocateMemory();
    initializeFields();
}

void BoltzmannSolver::allocateMemory() {
    size_t grid_size = nx_ * ny_ * nz_;
    hipMalloc(&d_f_distribution, grid_size * 19 * sizeof(float));
    hipMalloc(&d_g_distribution, grid_size * 7 * sizeof(float));
    hipMalloc(&d_density, grid_size * sizeof(float));
    hipMalloc(&d_velocity_x, grid_size * sizeof(float));
    hipMalloc(&d_velocity_y, grid_size * sizeof(float));
    hipMalloc(&d_velocity_z, grid_size * sizeof(float));
    hipMalloc(&d_temperature, grid_size * sizeof(float));
    hipMalloc(&d_tau_f, grid_size * sizeof(float));
    hipMalloc(&d_tau_t, grid_size * sizeof(float));
    hipMalloc(&d_vorticity_x, grid_size * sizeof(float));
    hipMalloc(&d_vorticity_y, grid_size * sizeof(float));
    hipMalloc(&d_vorticity_z, grid_size * sizeof(float));
    hipMalloc(&d_h_distribution, grid_size * 7 * sizeof(float));
    hipMalloc(&d_scalar_vorticity, grid_size * sizeof(float));
    
    // Allocate external force memory
    hipMalloc(&d_force_x, grid_size * sizeof(float));
    hipMalloc(&d_force_y, grid_size * sizeof(float));
    hipMalloc(&d_force_z, grid_size * sizeof(float));
    
    h_density = new float[grid_size];
    h_temperature = new float[grid_size];
    h_vel_.resize(grid_size * 3);
    h_rho_.resize(grid_size);
    h_tau_f_.resize(grid_size);
    h_tau_t_.resize(grid_size);
    h_vorticity_.resize(grid_size * 3);
    h_scalar_vorticity_.resize(grid_size);
    h_force_.resize(grid_size * 3);
}

void BoltzmannSolver::freeMemory() {
    hipFree(d_f_distribution);
    hipFree(d_g_distribution);
    hipFree(d_density);
    hipFree(d_velocity_x);
    hipFree(d_velocity_y);
    hipFree(d_velocity_z);
    hipFree(d_temperature);
    hipFree(d_tau_f);
    hipFree(d_tau_t);
    hipFree(d_vorticity_x);
    hipFree(d_vorticity_y);
    hipFree(d_vorticity_z);
    hipFree(d_h_distribution);
    hipFree(d_scalar_vorticity);
    
    // Free external force memory
    hipFree(d_force_x);
    hipFree(d_force_y);
    hipFree(d_force_z);
    
    delete[] h_density;
    delete[] h_temperature;
}

void BoltzmannSolver::initializeFields() {
    size_t grid_size = nx_ * ny_ * nz_;
    std::vector<float> initial_density(grid_size, 0.00f);
    std::vector<float> initial_velocity(grid_size * 3, 0.0f);
    std::vector<float> initial_f_distribution(grid_size * 19, 0.0f);
    std::vector<float> initial_g_distribution(grid_size * 7, 0.0f);
    std::vector<float> initial_tau_f(grid_size, init_params_.tau_f);
    std::vector<float> initial_tau_t(grid_size, init_params_.tau_t);
    std::vector<float> initial_temperature(grid_size, init_params_.temperature);

    hipMemcpy(d_f_distribution, initial_f_distribution.data(), grid_size * 19 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_g_distribution, initial_g_distribution.data(), grid_size * 7 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_density, initial_density.data(), grid_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity_x, initial_velocity.data(), grid_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity_y, initial_velocity.data(), grid_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity_z, initial_velocity.data(), grid_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_temperature, initial_temperature.data(), grid_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tau_f, initial_tau_f.data(), grid_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tau_t, initial_tau_t.data(), grid_size * sizeof(float), hipMemcpyHostToDevice);
    
    memcpy(h_density, initial_density.data(), grid_size * sizeof(float));
    memcpy(h_temperature, initial_temperature.data(), grid_size * sizeof(float));
    h_tau_f_ = initial_tau_f;
    h_tau_t_ = initial_tau_t;
}

void BoltzmannSolver::streamFluid() {
    dim3 block(8, 8, 8);
    dim3 grid((nx_ + block.x - 1) / block.x,
              (ny_ + block.y - 1) / block.y,
              (nz_ + block.z - 1) / block.z);
    
    // Allocate temporary buffer
    float* d_f_new;
    hipMalloc(&d_f_new, nx_ * ny_ * nz_ * 19 * sizeof(float));
        
    // Execute streaming kernel
    streamingKernel<<<grid, block>>>(d_f_distribution, d_f_new, nx_, ny_, nz_);
    hipDeviceSynchronize();
    
    // Copy results
    hipMemcpy(d_f_distribution, d_f_new, nx_ * ny_ * nz_ * 19 * sizeof(float), hipMemcpyDeviceToDevice);
    
    // Free temporary buffer
    hipFree(d_f_new);
}

void BoltzmannSolver::streamTemperature() {
    dim3 block(8, 8, 8);
    dim3 grid((nx_ + block.x - 1) / block.x,
              (ny_ + block.y - 1) / block.y,
              (nz_ + block.z - 1) / block.z);
    
    float* d_g_new;
    hipMalloc(&d_g_new, nx_ * ny_ * nz_ * 7 * sizeof(float));
    hipMemset(d_g_new, 0, nx_ * ny_ * nz_ * 7 * sizeof(float));
    int bc_type = static_cast<int>(init_params_.temperature_bc_type);
    float dirichlet_temp = init_params_.dirichlet_temperature;
    temperatureStreamingKernel<<<grid, block>>>(d_g_distribution, d_g_new, nx_, ny_, nz_, bc_type, dirichlet_temp);
    hipDeviceSynchronize();
    hipMemcpy(d_g_distribution, d_g_new, nx_ * ny_ * nz_ * 7 * sizeof(float), hipMemcpyDeviceToDevice);
    hipFree(d_g_new);
}

void BoltzmannSolver::collideFluid() {
    int grid_size = nx_ * ny_ * nz_;
    int block_size = 256;
    int num_blocks = (grid_size + block_size - 1) / block_size;
    fluidCollisionKernel<<<num_blocks, block_size>>>(d_f_distribution, d_density, d_velocity_x, d_velocity_y, d_velocity_z, d_tau_f, d_force_x, d_force_y, d_force_z, nx_, ny_, nz_, current_step_, init_params_.velocity_limit, init_params_.tau_rand_factor);
    hipDeviceSynchronize();
}

void BoltzmannSolver::collideTemperature() {
    int grid_size = nx_ * ny_ * nz_;
    int block_size = 256;
    int num_blocks = (grid_size + block_size - 1) / block_size;
    temperatureCollisionKernel<<<num_blocks, block_size>>>(d_g_distribution, d_temperature, d_velocity_x, d_velocity_y, d_velocity_z, d_tau_t, nx_, ny_, nz_, init_params_.velocity_limit);
    hipDeviceSynchronize();
}

void BoltzmannSolver::updateTemperature() {
    int grid_size = nx_ * ny_ * nz_;
    int block_size = 256;
    int num_blocks = (grid_size + block_size - 1) / block_size;
    updateTemperatureKernel<<<num_blocks, block_size>>>(d_g_distribution, d_temperature, nx_, ny_, nz_);
    hipDeviceSynchronize();
}

void BoltzmannSolver::injectSmokeSource() {
    
    dim3 block(8, 8, 8);
    dim3 grid((nx_ + block.x - 1) / block.x,
              (ny_ + block.y - 1) / block.y,
              (nz_ + block.z - 1) / block.z);

    injectSmokeSourceKernel<<<grid, block>>>(d_f_distribution, d_g_distribution, d_density, d_temperature,
                                            nx_, ny_, nz_, init_params_.source_radius, init_params_.source_density,
                                            init_params_.source_temperature, init_params_.source_injection_rate, current_step_);
    hipDeviceSynchronize();
}

void BoltzmannSolver::updateMacroscopic() {
    dim3 block(8, 8, 8);
    dim3 grid((nx_ + block.x - 1) / block.x,
              (ny_ + block.y - 1) / block.y,
              (nz_ + block.z - 1) / block.z);
    
    // Execute macroscopic quantity calculation kernel
    calculateMacroKernel<<<grid, block>>>(d_f_distribution, d_density, d_velocity_x, d_velocity_y, d_velocity_z, nx_, ny_, nz_);
    hipDeviceSynchronize();
}

void BoltzmannSolver::calculateVorticity() {
    dim3 block(8, 8, 8);
    dim3 grid((nx_ + block.x - 1) / block.x,
              (ny_ + block.y - 1) / block.y,
              (nz_ + block.z - 1) / block.z);
    
    calculateVorticityKernel<<<grid, block>>>(d_velocity_x, d_velocity_y, d_velocity_z, 
                                              d_vorticity_x, d_vorticity_y, d_vorticity_z, 
                                              nx_, ny_, nz_);
    hipDeviceSynchronize();
}

void BoltzmannSolver::streamVorticity() {
    dim3 block(8, 8, 8);
    dim3 grid((nx_ + block.x - 1) / block.x,
              (ny_ + block.y - 1) / block.y,
              (nz_ + block.z - 1) / block.z);
    
    float* d_h_new;
    hipMalloc(&d_h_new, nx_ * ny_ * nz_ * 7 * sizeof(float));
    hipMemset(d_h_new, 0, nx_ * ny_ * nz_ * 7 * sizeof(float));
    
    vorticityStreamingKernel<<<grid, block>>>(d_h_distribution, d_h_new, nx_, ny_, nz_);
    hipDeviceSynchronize();
    
    hipMemcpy(d_h_distribution, d_h_new, nx_ * ny_ * nz_ * 7 * sizeof(float), hipMemcpyDeviceToDevice);
    hipFree(d_h_new);
}

void BoltzmannSolver::collideVorticity() {
    int grid_size = nx_ * ny_ * nz_;
    int block_size = 256;
    int num_blocks = (grid_size + block_size - 1) / block_size;
    
    float lambda = 0.1f; // Vorticity constraint parameter
    vorticityCollisionKernel<<<num_blocks, block_size>>>(d_h_distribution, d_scalar_vorticity, 
                                                         d_velocity_x, d_velocity_y, d_velocity_z,
                                                         d_temperature, d_vorticity_x, d_vorticity_y, d_vorticity_z,
                                                         nx_, ny_, nz_, init_params_.beta, lambda);
    hipDeviceSynchronize();
}

void BoltzmannSolver::updateScalarVorticity() {
    int grid_size = nx_ * ny_ * nz_;
    int block_size = 256;
    int num_blocks = (grid_size + block_size - 1) / block_size;
    
    updateScalarVorticityKernel<<<num_blocks, block_size>>>(d_h_distribution, d_scalar_vorticity, nx_, ny_, nz_);
    hipDeviceSynchronize();
}

void BoltzmannSolver::updateExternalForces() {
    int grid_size = nx_ * ny_ * nz_;
    int block_size = 256;
    int num_blocks = (grid_size + block_size - 1) / block_size;
    
    // Lambda parameter for vorticity constraint force
    float lambda = 0.1f;
    
    updateExternalForcesKernel<<<num_blocks, block_size>>>(d_force_x, d_force_y, d_force_z, 
                                                          d_temperature, d_vorticity_x, d_vorticity_y, d_vorticity_z,
                                                          nx_, ny_, nz_, init_params_.beta, lambda, current_step_);
    hipDeviceSynchronize();
}

void BoltzmannSolver::copyToHost() {
    // Data transfer from GPU to CPU
    hipMemcpy(h_density, d_density, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_temperature, d_temperature, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    
    // Transfer velocity vectors
    std::vector<float> temp_vel_x(nx_ * ny_ * nz_);
    std::vector<float> temp_vel_y(nx_ * ny_ * nz_);
    std::vector<float> temp_vel_z(nx_ * ny_ * nz_);
    
    hipMemcpy(temp_vel_x.data(), d_velocity_x, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(temp_vel_y.data(), d_velocity_y, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(temp_vel_z.data(), d_velocity_z, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    
    // Store velocity vectors in h_vel_
    h_vel_.resize(nx_ * ny_ * nz_ * 3);
    for (int i = 0; i < nx_ * ny_ * nz_; i++) {
        h_vel_[i] = temp_vel_x[i];
        h_vel_[i + nx_ * ny_ * nz_] = temp_vel_y[i];
        h_vel_[i + 2 * nx_ * ny_ * nz_] = temp_vel_z[i];
    }
    
    // Transfer vorticity vectors
    std::vector<float> temp_vort_x(nx_ * ny_ * nz_);
    std::vector<float> temp_vort_y(nx_ * ny_ * nz_);
    std::vector<float> temp_vort_z(nx_ * ny_ * nz_);
    
    hipMemcpy(temp_vort_x.data(), d_vorticity_x, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(temp_vort_y.data(), d_vorticity_y, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(temp_vort_z.data(), d_vorticity_z, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    
    // Store vorticity vectors in h_vorticity_
    h_vorticity_.resize(nx_ * ny_ * nz_ * 3);
    for (int i = 0; i < nx_ * ny_ * nz_; i++) {
        h_vorticity_[i] = temp_vort_x[i];
        h_vorticity_[i + nx_ * ny_ * nz_] = temp_vort_y[i];
        h_vorticity_[i + 2 * nx_ * ny_ * nz_] = temp_vort_z[i];
    }
    
    // Transfer scalar vorticity
    hipMemcpy(h_scalar_vorticity_.data(), d_scalar_vorticity, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    
    // Transfer external forces
    std::vector<float> temp_force_x(nx_ * ny_ * nz_);
    std::vector<float> temp_force_y(nx_ * ny_ * nz_);
    std::vector<float> temp_force_z(nx_ * ny_ * nz_);
    
    hipMemcpy(temp_force_x.data(), d_force_x, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(temp_force_y.data(), d_force_y, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(temp_force_z.data(), d_force_z, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyDeviceToHost);
    
    // Store force vectors in h_force_
    h_force_.resize(nx_ * ny_ * nz_ * 3);
    for (int i = 0; i < nx_ * ny_ * nz_; i++) {
        h_force_[i] = temp_force_x[i];
        h_force_[i + nx_ * ny_ * nz_] = temp_force_y[i];
        h_force_[i + 2 * nx_ * ny_ * nz_] = temp_force_z[i];
    }
    
    // Output debug information
    float max_density = 0.0f;
    float avg_density = 0.0f;
    int active_voxels = 0;
    
    for (int i = 0; i < nx_ * ny_ * nz_; i++) {
        if (h_density[i] > 0.1f) {
            active_voxels++;
            max_density = std::max(max_density, h_density[i]);
            avg_density += h_density[i];
        }
    }
    
    if (active_voxels > 0) {
        avg_density /= active_voxels;
    }
}

void BoltzmannSolver::simulate(float dt, int steps) {
    float period = 30.0f;
    float omega = 2.0f * 3.14159265358979323846f / period;
    for (int step = 0; step < steps; ++step) {
        hipMemcpy(d_density, h_density, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_temperature, h_temperature, nx_ * ny_ * nz_ * sizeof(float), hipMemcpyHostToDevice);

        // Calculate vorticity for VPLBM
        calculateVorticity();
        
        // Update external forces
        updateExternalForces();

        // Fluid simulation steps
        collideFluid();
        collideTemperature();
        collideVorticity();

        streamFluid();
        streamTemperature();
        streamVorticity();

        updateMacroscopic();
        updateTemperature();
        updateScalarVorticity();

        if(init_params_.continuous_source && current_step_ % init_params_.source_injection_interval == 0) {
            injectSmokeSource();
        }

        copyToHost();

        // Display progress
        std::cout << "\rStep " << current_step_ 
                  << " | Active voxels: " << countActiveVoxels() 
                  << " | Max density: " << getMaxDensity() 
                  << " | Avg density: " << getAverageDensity() 
                  << " | Max temperature: " << getMaxTemperature()
                  << " | Avg temperature: " << getAverageTemperature()
                  << " | Avg velocity: " << getAverageVelocity()
                  << " | Max velocity: " << getMaxVelocity()
                  << std::flush;

        current_step_++;
    }
}

BoltzmannSolver::~BoltzmannSolver() {
    freeMemory();
}

// Implementation of statistical functions
int BoltzmannSolver::countActiveVoxels() const {
    int count = 0;
    int grid_size = nx_ * ny_ * nz_;
    for (int i = 0; i < grid_size; i++) {
        if (h_density[i] > 0.001f) {
            count++;
        }
    }
    return count;
}

float BoltzmannSolver::getMaxDensity() const {
    float max_density = 0.0f;
    int grid_size = nx_ * ny_ * nz_;
    for (int i = 0; i < grid_size; i++) {
        max_density = std::max(max_density, h_density[i]);
    }
    return max_density;
}

float BoltzmannSolver::getAverageDensity() const {
    float total_density = 0.0f;
    int active_count = 0;
    int grid_size = nx_ * ny_ * nz_;
    for (int i = 0; i < grid_size; i++) {
        if (h_density[i] > 0.001f) {
            total_density += h_density[i];
            active_count++;
        }
    }
    return active_count > 0 ? total_density / active_count : 0.0f;
}

float BoltzmannSolver::getMaxTemperature() const {
    float max_temp = 0.0f;
    int grid_size = nx_ * ny_ * nz_;
    for (int i = 0; i < grid_size; i++) {
        max_temp = std::max(max_temp, h_temperature[i]);
    }
    return max_temp;
}

float BoltzmannSolver::getAverageTemperature() const {
    float total_temp = 0.0f;
    int active_count = 0;
    int grid_size = nx_ * ny_ * nz_;
    for (int i = 0; i < grid_size; i++) {
        if (h_density[i] > 0.001f) {
            total_temp += h_temperature[i];
            active_count++;
        }
    }
    return active_count > 0 ? total_temp / active_count : 0.0f;
}

float BoltzmannSolver::getAverageVelocity() const {
    float total_velocity = 0.0f;
    int active_count = 0;
    int grid_size = nx_ * ny_ * nz_;
    for (int i = 0; i < grid_size; i++) {
        if (h_vel_[3*i] > 0.001f) {
            float vx = h_vel_[3*i];
            float vy = h_vel_[3*i + 1];
            float vz = h_vel_[3*i + 2];
            float velocity_magnitude = sqrtf(vx*vx + vy*vy + vz*vz);
            total_velocity += velocity_magnitude;
            active_count++;
        }
    }
    return active_count > 0 ? total_velocity / active_count : 0.0f;
}

float BoltzmannSolver::getMaxVelocity() const {
    float max_velocity = 0.0f;
    int grid_size = nx_ * ny_ * nz_;
    for (int i = 0; i < grid_size; i++) {
        float vx = h_vel_[3*i];
        float vy = h_vel_[3*i + 1];
        float vz = h_vel_[3*i + 2];
        float velocity_magnitude = sqrtf(vx*vx + vy*vy + vz*vz);
        max_velocity = std::max(max_velocity, velocity_magnitude);
    }
    return max_velocity;
}
